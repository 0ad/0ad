#include "hip/hip_runtime.h"
// Copyright NVIDIA Corporation 2007 -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "CudaMath.h"

#define TW 16
#define TH 16

#define THREAD_COUNT 		(TW * TH)

#define MAX_KERNEL_WIDTH	32

#define KW 4



#if __DEVICE_EMULATION__
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif

#define TN            256
#define WARP_COUNT    (TN / 32)
#define HWARP_COUNT   (TN / 16)

// Window size
#define WS            20



struct WrapClamp
{
	int operator()(int i, int h)
	{
		i = min(max(i, 0), h-1);
	}
};

struct WrapRepeat
{
	int operator()(int i, int h)
	{
		i = abs(i) % h;	// :( Non power of two!
	}
};

struct WrapMirror
{
	int operator()(int i, int h)
	{
		i = abs(i);
		while (i >= h) i = 2 * w - i - 2;
	}
};


// Vertical convolution filter that processes vertical strips.
__global__ void convolveStrip(float * d_channel, float * d_kernel, int width, int height)
{
	__shared__ float s_kernel[32 * WS];

	// Preload kernel in shared memory.
	for (int i = 0; i < 32 * WS / TN; i++)
	{
		int idx = i * TN + tid;
		if (idx < 32 * WS) s_kernel[idx] = d_kernel[idx];
	}

	__shared__ float s_strip[32 * WS];	// TN/32
	
	int wid = tid / 32 - WS/2;

	Mirror wrap;
	int row = wrap(wid);

	// Preload image block.
	for (int i = 0; i < 32 * WS / TN; i++)
	{
	}

	// @@ Apply kernel to TN/32 rows.

	// @@ Load 


}






__constant__ float inputGamma, outputInverseGamma;
__constant__ float kernel[MAX_KERNEL_WIDTH];

// Use texture to access input?
// That's the most simple approach.

texture<> image;

////////////////////////////////////////////////////////////////////////////////
// Combined convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convolve(float4 * output)
{
	// @@ Use morton order to assing threads.
	int x = threadIdx.x;
	int y = threadIdx.y;
	
	float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	
	// texture coordinate.
	int2 t;
	t.x = 2 * (blockIdx.x * TW + x) - HW;
	t.y = blockIdx.y * TH + y;
	
	// @@ We might want to loop and process strips, to reuse the results of the horizontal convolutions.
	
	// Horizontal convolution. @@ Unroll loops.
	for (int e = HW; e > 0; e--)
	{
		t.x++;
		float w = kernel[e-1];
		color += w * tex2D(image, tc);
	}
	
	for (int e = 0; e < HW; e++)
	{
		t.x++;
		float w = kernel[e];
		color += w * tex2D(image, tc);
	}
	
	// Write color to shared memory.
	__shared__ float tile[4 * THREAD_COUNT];

	int tileIdx = y * TW + x;
	tile[tileIdx + 0 * THREAD_COUNT] = color.x;
	tile[tileIdx + 1 * THREAD_COUNT] = color.y;
	tile[tileIdx + 2 * THREAD_COUNT] = color.z;
	tile[tileIdx + 3 * THREAD_COUNT] = color.w;

	__syncthreads();
	
	// tile coordinate.
	t.x = x;
	t.y = y - HW;
	
	// Vertical convolution. @@ Unroll loops.
	for (int i = HW; i > 0; i--)
	{
		float w = kernel[i-1];
		
		t.y++;
		int idx = t.y * TW + t.x;
		
		color.x += w * tile[idx + 0 * THREAD_COUNT];
		color.y += w * tile[idx + 1 * THREAD_COUNT];
		color.z += w * tile[idx + 2 * THREAD_COUNT];
		color.w += w * tile[idx + 3 * THREAD_COUNT];
	}
	
	for (int i = 0; i < HW; i++)
	{
		float w = kernel[i];
		
		t.y++;
		int idx = t.y * TW + t.x;
		
		color.x += w * tile[idx + 0 * THREAD_COUNT];
		color.y += w * tile[idx + 1 * THREAD_COUNT];
		color.z += w * tile[idx + 2 * THREAD_COUNT];
		color.w += w * tile[idx + 3 * THREAD_COUNT];
	}	
	
	it (x < w && y < h)
	{
		// @@ Prevent unaligned writes.
		
		output[y * w + h] = color;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Monophase X convolution filter
////////////////////////////////////////////////////////////////////////////////

__device__ void convolveY()
{

}


////////////////////////////////////////////////////////////////////////////////
// Mipmap convolution filter
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// Gamma correction
////////////////////////////////////////////////////////////////////////////////

/*
__device__ float toLinear(float f, float gamma = 2.2f)
{
	return __pow(f, gamma);
}

__device__ float toGamma(float f, float gamma = 2.2f)
{
	return pow(f, 1.0f / gamma);
}
*/




////////////////////////////////////////////////////////////////////////////////
// Setup kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void setupConvolveKernel(const float * k, int w)
{
	w = min(w, MAX_KERNEL_WIDTH);
	hipMemcpyToSymbol(HIP_SYMBOL(kernel), k, sizeof(float) * w, 0);
}


////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////





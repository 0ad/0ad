#include "hip/hip_runtime.h"
// Copyright NVIDIA Corporation 2007 -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "CudaMath.h"

#include "../SingleColorLookup.h"

#define NUM_THREADS 64		// Number of threads per block.

#if __DEVICE_EMULATION__
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif

typedef unsigned char uchar;
typedef unsigned short ushort;
typedef unsigned int uint;

template <class T> 
__device__ inline void swap(T & a, T & b)
{
	T tmp = a;
	a = b;
	b = tmp;
}

__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };
__constant__ float3 kColorMetricSqr = { 1.0f, 1.0f, 1.0f };



////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float * values, int * cmp)
{
	int tid = threadIdx.x;

#if 1
	cmp[tid] = (values[0] < values[tid]);
	cmp[tid] += (values[1] < values[tid]);
	cmp[tid] += (values[2] < values[tid]);
	cmp[tid] += (values[3] < values[tid]);
	cmp[tid] += (values[4] < values[tid]);
	cmp[tid] += (values[5] < values[tid]);
	cmp[tid] += (values[6] < values[tid]);
	cmp[tid] += (values[7] < values[tid]);
	cmp[tid] += (values[8] < values[tid]);
	cmp[tid] += (values[9] < values[tid]);
	cmp[tid] += (values[10] < values[tid]);
	cmp[tid] += (values[11] < values[tid]);
	cmp[tid] += (values[12] < values[tid]);
	cmp[tid] += (values[13] < values[tid]);
	cmp[tid] += (values[14] < values[tid]);
	cmp[tid] += (values[15] < values[tid]);
	
	// Resolve elements with the same index.
	if (tid > 0 && cmp[tid] == cmp[0]) ++cmp[tid];
	if (tid > 1 && cmp[tid] == cmp[1]) ++cmp[tid];
	if (tid > 2 && cmp[tid] == cmp[2]) ++cmp[tid];
	if (tid > 3 && cmp[tid] == cmp[3]) ++cmp[tid];
	if (tid > 4 && cmp[tid] == cmp[4]) ++cmp[tid];
	if (tid > 5 && cmp[tid] == cmp[5]) ++cmp[tid];
	if (tid > 6 && cmp[tid] == cmp[6]) ++cmp[tid];
	if (tid > 7 && cmp[tid] == cmp[7]) ++cmp[tid];
	if (tid > 8 && cmp[tid] == cmp[8]) ++cmp[tid];
	if (tid > 9 && cmp[tid] == cmp[9]) ++cmp[tid];
	if (tid > 10 && cmp[tid] == cmp[10]) ++cmp[tid];
	if (tid > 11 && cmp[tid] == cmp[11]) ++cmp[tid];
	if (tid > 12 && cmp[tid] == cmp[12]) ++cmp[tid];
	if (tid > 13 && cmp[tid] == cmp[13]) ++cmp[tid];
	if (tid > 14 && cmp[tid] == cmp[14]) ++cmp[tid];
#else

	cmp[tid] = 0;

	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		cmp[tid] += (values[i] < values[tid]);
	}

	// Resolve elements with the same index.
	#pragma unroll
	for (int i = 0; i < 15; i++)
	{
		if (tid > 0 && cmp[tid] == cmp[i]) ++cmp[tid];
	}
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
		
		colors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		
		// No need to synchronize, 16 < warp size.
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float3 axis = bestFitLine(colors, sums[0], kColorMetric);
		
		*sameColor = (axis == make_float3(0, 0, 0));
		
		dps[idx] = dot(colors[idx], axis);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		colors[xrefs[idx]] = tmp;
	}
}

__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], float weights[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float3 rawColors[16];
	__shared__ float dps[16];

	if (idx < 16)
	{
		// Read color and copy to shared mem.
		uint c = image[(bid) * 16 + idx];
		
		rawColors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		rawColors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		rawColors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		weights[idx] = (((c >> 24) & 0xFF) + 1) * (1.0f / 256.0f);
		
		colors[idx] = rawColors[idx] * weights[idx];

		
		// No need to synchronize, 16 < warp size.
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif

		// Sort colors along the best fit line.
		colorSums(colors, sums);
		float3 axis = bestFitLine(colors, sums[0], kColorMetric);

		*sameColor = (axis == make_float3(0, 0, 0));

		// Single color compressor needs unweighted colors.
		if (*sameColor) colors[idx] = rawColors[idx];

		dps[idx] = dot(rawColors[idx], axis);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		sortColors(dps, xrefs);
		
		float3 tmp = colors[idx];
		colors[xrefs[idx]] = tmp;
		
		float w = weights[idx];
		weights[xrefs[idx]] = w;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand565(float3 v, ushort * w)
{
	v.x = rintf(__saturatef(v.x) * 31.0f);
	v.y = rintf(__saturatef(v.y) * 63.0f);
	v.z = rintf(__saturatef(v.z) * 31.0f);
	*w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
	v.x *= 0.03227752766457f; // approximate integer bit expansion.
	v.y *= 0.01583151765563f;
	v.z *= 0.03227752766457f;
	return v;
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
__device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		float beta = (bits & 1);
		if (bits & 2) beta = (1 + beta) / 3.0f;
		float alpha = 1.0f - beta;
		
		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * colors[i];
		betax_sum += beta * colors[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		float beta = (bits & 1);
		if (bits & 2) beta = 0.5f;
		float alpha = 1.0f - beta;
	
		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * colors[i];
		betax_sum += beta * colors[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

__constant__ const float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ const float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const uint prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const uint prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

__device__ float evalPermutation4(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		alphax_sum += alphaTable4[bits & 3] * colors[i];
		akku += prods4[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float3 betax_sum = 9.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 9.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
	uint akku = 0;

	// Compute alpha & beta for this permutation.
	#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		alphax_sum += alphaTable3[bits & 3] * colors[i];
		akku += prods3[bits & 3];
	}

	float alpha2_sum = float(akku >> 16);
	float beta2_sum = float((akku >> 8) & 0xff);
	float alphabeta_sum = float(akku & 0xff);
	float3 betax_sum = 4.0f * color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return (1.0f / 4.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation4(const float3 * colors, const float * weights, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);
		
		float beta = (bits & 1);
		if (bits & 2) beta = (1 + beta) / 3.0f;
		float alpha = 1.0f - beta;
		
		alpha2_sum += alpha * alpha * weights[i];
		beta2_sum += beta * beta * weights[i];
		alphabeta_sum += alpha * beta * weights[i];
		alphax_sum += alpha * colors[i];
	}

	float3 betax_sum = color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
	
	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}

/*
__device__ float evalPermutation3(const float3 * colors, const float * weights, uint permutation, ushort * start, ushort * end)
{
	// Compute endpoints using least squares.
	float alpha2_sum = 0.0f;
	float beta2_sum = 0.0f;
	float alphabeta_sum = 0.0f;
	float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

	// Compute alpha & beta for this permutation.
	for (int i = 0; i < 16; i++)
	{
		const uint bits = permutation >> (2*i);

		float beta = (bits & 1);
		if (bits & 2) beta = 0.5f;
		float alpha = 1.0f - beta;

		alpha2_sum += alpha * alpha * weights[i];
		beta2_sum += beta * beta * weights[i];
		alphabeta_sum += alpha * beta * weights[i];
		alphax_sum += alpha * colors[i];
	}

	float3 betax_sum = color_sum - alphax_sum;

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

	// Round a, b to the closest 5-6-5 color and expand...
	a = roundAndExpand565(a, start);
	b = roundAndExpand565(b, end);

	// compute the error
	float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

	return dot(e, kColorMetricSqr);
}
*/


////////////////////////////////////////////////////////////////////////////////
// Evaluate all permutations
////////////////////////////////////////////////////////////////////////////////
__device__ void evalAllPermutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];

	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;
				
		float error = evalPermutation4(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}

/*
__device__ void evalAllPermutations(const float3 * colors, const float * weights, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;

		float error = evalPermutation4(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}
*/

__device__ void evalLevel4Permutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];

		float error = evalPermutation4(colors, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	errors[idx] = bestError;
}

__device__ void evalLevel4Permutations(const float3 * colors, const float * weights, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];

		float error = evalPermutation4(colors, weights, colorSum, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	errors[idx] = bestError;
}


////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float * errors)
{
	const int idx = threadIdx.x;

	__shared__ int indices[NUM_THREADS];
	indices[idx] = idx;

#if __DEVICE_EMULATION__
	for(int d = NUM_THREADS/2; d > 0; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

#else
	for(int d = NUM_THREADS/2; d > 32; d >>= 1)
	{
		__syncthreads();
		
		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

	__syncthreads();

	// unroll last 6 iterations
	if (idx < 32)
	{
		if (errors[idx + 32] < errors[idx]) {
			errors[idx] = errors[idx + 32];
			indices[idx] = indices[idx + 32];
		}
		if (errors[idx + 16] < errors[idx]) {
			errors[idx] = errors[idx + 16];
			indices[idx] = indices[idx + 16];
		}
		if (errors[idx + 8] < errors[idx]) {
			errors[idx] = errors[idx + 8];
			indices[idx] = indices[idx + 8];
		}
		if (errors[idx + 4] < errors[idx]) {
			errors[idx] = errors[idx + 4];
			indices[idx] = indices[idx + 4];
		}
		if (errors[idx + 2] < errors[idx]) {
			errors[idx] = errors[idx + 2];
			indices[idx] = indices[idx + 2];
		}
		if (errors[idx + 1] < errors[idx]) {
			errors[idx] = errors[idx + 1];
			indices[idx] = indices[idx + 1];
		}
	}
#endif

	__syncthreads();

	return indices[0];
}


////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
	const int bid = blockIdx.x;

	if (start == end)
	{
		permutation = 0;
	}
	
	// Reorder permutation.
	uint indices = 0;
	for(int i = 0; i < 16; i++)
	{
		int ref = xrefs[i];
		indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
	}
	
	// Write endpoints.
	result[bid].x = (end << 16) | start;
	
	// Write palette indices.
	result[bid].y = indices;
}

__device__ void saveSingleColorBlockDXT1(float3 color, uint2 * result)
{
	const int bid = blockIdx.x;

	int r = color.x * 255;
	int g = color.y * 255;
	int b = color.z * 255;

	ushort color0 = (OMatch5[r][0] << 11) | (OMatch6[g][0] << 5) | OMatch5[b][0];
	ushort color1 = (OMatch5[r][1] << 11) | (OMatch6[g][1] << 5) | OMatch5[b][1];

	if (color0 < color1)
	{
		result[bid].x = (color0 << 16) | color1;
		result[bid].y = 0xffffffff;
	}
	else
	{
		result[bid].x = (color1 << 16) | color0;
		result[bid].y = 0xaaaaaaaa;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compressDXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, xrefs, &sameColor);

	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}

__global__ void compressLevel4DXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, xrefs, &sameColor);

	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];

	evalLevel4Permutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	const int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}

__global__ void compressWeightedDXT1(const uint * permutations, const uint * image, uint2 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	__shared__ int sameColor;
	
	loadColorBlock(image, colors, sums, weights, xrefs, &sameColor);
	
	__syncthreads();

	if (sameColor)
	{
		if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
		return;
	}

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];
	
	evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
	}
}


/*
__device__ float computeError(const float weights[16], uchar a0, uchar a1)
{
	float palette[6];
	palette[0] = (6.0f/7.0f * a0 + 1.0f/7.0f * a1);
	palette[1] = (5.0f/7.0f * a0 + 2.0f/7.0f * a1);
	palette[2] = (4.0f/7.0f * a0 + 3.0f/7.0f * a1);
	palette[3] = (3.0f/7.0f * a0 + 4.0f/7.0f * a1);
	palette[4] = (2.0f/7.0f * a0 + 5.0f/7.0f * a1);
	palette[5] = (1.0f/7.0f * a0 + 6.0f/7.0f * a1);

	float total = 0.0f;

	for (uint i = 0; i < 16; i++)
	{
		float alpha = weights[i];

		float error = a0 - alpha;
		error = min(error, palette[0] - alpha);
		error = min(error, palette[1] - alpha);
		error = min(error, palette[2] - alpha);
		error = min(error, palette[3] - alpha);
		error = min(error, palette[4] - alpha);
		error = min(error, palette[5] - alpha);
		error = min(error, a1 - alpha);
		
		total += error;
	}
	
	return total;
}

inline __device__ uchar roundAndExpand(float a)
{
	return rintf(__saturatef(a) * 255.0f);
}
*/
/*
__device__ void optimizeAlpha8(const float alphas[16], uchar & a0, uchar & a1)
{
	float alpha2_sum = 0;
	float beta2_sum = 0;
	float alphabeta_sum = 0;
	float alphax_sum = 0;
	float betax_sum = 0;

	for (int i = 0; i < 16; i++)
	{
		uint idx = index[i];
		float alpha;
		if (idx < 2) alpha = 1.0f - idx;
		else alpha = (8.0f - idx) / 7.0f;
		
		float beta = 1 - alpha;

		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * alphas[i];
		betax_sum += beta * alphas[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

	a0 = roundAndExpand8(a);
	a1 = roundAndExpand8(b);
}
*/
/*
__device__ void compressAlpha(const float alphas[16], uint4 * result)
{
	const int tid = threadIdx.x;
	
	// Compress alpha block!
	// Brute force approach:
	// Try all color pairs: 256*256/2 = 32768, 32768/64 = 512 iterations?

	// Determine min & max alphas

	float A0, A1;

	if (tid < 16)
	{
		__shared__ uint s_alphas[16];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^1]);
		A0 = s_alphas[tid];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^1]);
		A1 = s_alphas[tid];
	}

	__syncthreads();

	int minIdx = 0;

	if (A1 - A0 > 8)
	{
		float bestError = FLT_MAX;

		// 64 threads -> 8x8
		// divide [A1-A0] in partitions.
		// test endpoints 
		
		for (int i = 0; i < 128; i++)
		{
			uint idx = (i * NUM_THREADS + tid) * 4;
			uchar a0 = idx & 255;
			uchar a1 = idx >> 8;
			
			float error = computeError(alphas, a0, a1);
			
			if (error < bestError)
			{
				bestError = error;
				A0 = a0;
				A1 = a1;
			}
		}
		
		__shared__ float errors[NUM_THREADS];
		errors[tid] = bestError;
		
		// Minimize error.
		minIdx = findMinError(errors);

	}

	if (minIdx == tid)
	{
		// @@ Compute indices.
	
		// @@ Write alpha block.
	}
}

__global__ void compressDXT5(const uint * permutations, const uint * image, uint4 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	
	loadColorBlock(image, colors, sums, weights, xrefs);
	
	__syncthreads();

	compressAlpha(weights, result);	

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];
	
	evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, (uint2 *)result);
	}
}
*/

////////////////////////////////////////////////////////////////////////////////
// Setup kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void setupCompressKernel(const float weights[3])
{
	// Set constants.
	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetric), weights, sizeof(float) * 3, 0);

	float weightsSqr[3];
	weightsSqr[0] = weights[0] * weights[0];
	weightsSqr[1] = weights[1] * weights[1];
	weightsSqr[2] = weights[2] * weights[2];

	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetricSqr), weightsSqr, sizeof(float) * 3, 0);
}


////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void compressKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressKernelDXT1_Level4(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressLevel4DXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressWeightedKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
	compressWeightedDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

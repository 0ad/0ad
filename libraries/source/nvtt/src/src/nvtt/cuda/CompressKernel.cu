#include "hip/hip_runtime.h"
// Copyright (c) 2009-2011 Ignacio Castano <castano@gmail.com>
// Copyright (c) 2007-2009 NVIDIA Corporation -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <math.h>
#include <float.h> // FLT_MAX

#include "CudaMath.h"


#define NUM_THREADS 64		// Number of threads per block.

typedef unsigned char uchar;
typedef unsigned short ushort;
typedef unsigned int uint;

template <class T> 
__device__ inline void swap(T & a, T & b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

__constant__ uchar OMatch5[256][2];
__constant__ uchar OMatch6[256][2];

__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };
__constant__ float3 kColorMetricSqr = { 1.0f, 1.0f, 1.0f };

// Some kernels read the input through texture.
texture<uchar4, 2, hipReadModeNormalizedFloat> tex;


////////////////////////////////////////////////////////////////////////////////
// Color helpers
////////////////////////////////////////////////////////////////////////////////

__device__ inline uint float_to_u8(float value)
{
    return min(max(__float2int_rn((255 * value + 0.5f) / (1.0f + 1.0f/255.0f)), 0), 255);
}

__device__ inline uint float_to_u6(float value)
{
    return min(max(__float2int_rn((63 * value + 0.5f) / (1.0f + 1.0f/63.0f)), 0), 63);
}

__device__ inline uint float_to_u5(float value)
{
    return min(max(__float2int_rn((31 * value + 0.5f) / (1.0f + 1.0f/31.0f)), 0), 31);
}

__device__ inline float u8_to_float(uint value)
{
    return __saturatef(__uint2float_rn(value) / 255.0f);
    //return (value) / 255.0f;
}

__device__ float3 color32ToFloat3(uint c)
{
    float3 color;
    color.z = u8_to_float((c >> 0) & 0xFF);
    color.y = u8_to_float((c >> 8) & 0xFF);
    color.x = u8_to_float((c >> 16) & 0xFF);
    return color;
}

__device__ int3 color16ToInt3(ushort c)
{
    int3 color;

    color.z = ((c >> 0) & 0x1F);
    color.z = (color.z << 3) | (color.z >> 2);

    color.y = ((c >> 5) & 0x3F);
    color.y = (color.y << 2) | (color.y >> 4);

    color.x = ((c >> 11) & 0x1F);
    color.x = (color.x << 3) | (color.x >> 2);
    
    return color;
}

__device__ float3 color16ToFloat3(ushort c)
{
    int3 color = color16ToInt3(c);
    return make_float3(color.x, color.y, color.z) * (1.0f / 255.0f);
}

__device__ int3 float3ToInt3(float3 c)
{
    return make_int3(c.x * 255, c.y * 255, c.z * 255);
}

__device__ float3 int3ToFloat3(int3 c)
{
    return make_float3(float_to_u8(c.x), float_to_u8(c.y), float_to_u8(c.z));
}


__device__ int colorDistance(int3 c0, int3 c1)
{
    int dx = c0.x-c1.x;
    int dy = c0.y-c1.y;
    int dz = c0.z-c1.z;
    return __mul24(dx, dx) + __mul24(dy, dy) + __mul24(dz, dz);
}


////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////


#if 0
__device__ inline uint float_to_u8(float value)
{
    //uint result;
    //asm("cvt.sat.rni.u8.f32 %0, %1;" : "=r" (result) : "f" (value));
    //return result;
    //return __float2uint_rn(__saturatef(value) * 255.0f);
    
    int result = __float2int_rn((255 * value + 0.5f) / (1.0f + 1.0f/255.0f));
    result = max(result, 0);
    result = min(result, 255);
    return result;
}

__device__ inline float u8_to_float(uint value)
{
    //float result;
    //asm("cvt.sat.rn.f32.u8 %0, %1;" : "=f" (result) : "r" (value)); // this is wrong!
    //return result;
    return __saturatef(__uint2float_rn(value) / 255.0f);
}

inline __device__ float3 roundAndExpand565(float3 v, ushort * w)
{
    uint x = float_to_u8(v.x) >> 3;
    uint y = float_to_u8(v.y) >> 2;
    uint z = float_to_u8(v.z) >> 3;
    *w = (x << 11) | (y << 5) | z;
    v.x = u8_to_float((x << 3) | (x >> 2));
    v.y = u8_to_float((y << 2) | (y >> 4));
    v.z = u8_to_float((z << 3) | (z >> 2));
//    v.x = u8_to_float(x) * 255.0f / 31.0f;
//    v.y = u8_to_float(y) * 255.0f / 63.0f;
//    v.z = u8_to_float(z) * 255.0f / 31.0f;
    return v;
}
#else

inline __device__ float3 roundAndExpand565(float3 v, ushort * w)
{
    uint x = __float2uint_rn(__saturatef(v.x) * 31.0f);
    uint y = __float2uint_rn(__saturatef(v.y) * 63.0f);
    uint z = __float2uint_rn(__saturatef(v.z) * 31.0f);

    //uint x = float_to_u5(v.x);
    //uint y = float_to_u6(v.y);
    //uint z = float_to_u5(v.z);

    *w = (x << 11) | (y << 5) | z;

    v.x = __uint2float_rn(x) * 1.0f / 31.0f;
    v.y = __uint2float_rn(y) * 1.0f / 63.0f;
    v.z = __uint2float_rn(z) * 1.0f / 31.0f;

    //v.x = u8_to_float((x << 3) | (x >> 2));
    //v.y = u8_to_float((y << 2) | (y >> 4));
    //v.z = u8_to_float((z << 3) | (z >> 2));

    return v;
}
#endif
inline __device__ float2 roundAndExpand56(float2 v, ushort * w)
{
    uint x = __float2uint_rn(__saturatef(v.x) * 31.0f);
    uint y = __float2uint_rn(__saturatef(v.y) * 63.0f);
    *w = (x << 11) | (y << 5);
    v.x = __uint2float_rn(x) * 1.0f / 31.0f;
    v.y = __uint2float_rn(y) * 1.0f / 63.0f;
    return v;
}

inline __device__ float2 roundAndExpand88(float2 v, ushort * w)
{
    uint x = __float2uint_rn(__saturatef(v.x) * 255.0f);
    uint y = __float2uint_rn(__saturatef(v.y) * 255.0f);
    *w = (x << 8) | y;
    v.x = __uint2float_rn(x) * 1.0f / 255.0f;
    v.y = __uint2float_rn(y) * 1.0f / 255.0f;
    return v;
}


////////////////////////////////////////////////////////////////////////////////
// Block errors
////////////////////////////////////////////////////////////////////////////////

__device__ float3 blockError4(const float3 * colors, uint permutation, float3 a, float3 b)
{
    float3 error = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;

        float3 diff = colors[i] - (a*alpha + b*beta);

        error += diff*diff;
    }

    return error;
}

__device__ float3 blockError4(const float3 * colors, uint permutation, ushort c0, ushort c1)
{
    float3 error = make_float3(0.0f, 0.0f, 0.0f);
    
    int3 color0 = color16ToInt3(c0);
    int3 color1 = color16ToInt3(c1);

    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        int beta = (bits & 1);
        if (bits & 2) beta = (1 + beta);
        float alpha = 3 - beta;

        int3 color;
        color.x = (color0.x * alpha + color1.x * beta) / 3;
        color.y = (color0.y * alpha + color1.y * beta) / 3;
        color.z = (color0.z * alpha + color1.z * beta) / 3;

        float3 diff = colors[i] - int3ToFloat3(color);

        error += diff*diff;
    }

    return error;
}


__device__ float3 blockError3(const float3 * colors, uint permutation, float3 a, float3 b)
{
    float3 error = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;

        float3 diff = colors[i] - (a*alpha + b*beta);

        error += diff*diff;
    }

    return error;
}


////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////

// @@ Experimental code to avoid duplicate colors for faster compression.
// We could first sort along the best fit line and only compare colors that have the same projection.
// The hardest part is to maintain the indices to map packed/sorted colors to the input colors.
// We also need to update several functions that assume the number of colors is fixed to 16.
// And compute different bit maps for the different color counts.
// This is a fairly high amount of work.
__device__ int packColors(float3 * values, float * weights, int * ranks)
{
    const int tid = threadIdx.x;

    __shared__ int count;
    count = 0;

    bool alive = true;

    // Append this
    for (int i = 0; i < 16; i++)
    {
        // One thread leads on each iteration.
        if (tid == i) {

            // If thread alive, then append element.
            if (alive) {
                values[count] = values[i];
                weights[count] = weights[i];
                count++;
            }

            // Otherwise update weight.
            else {
                weights[ranks[i]] += weights[i];
            }
        }

        // Kill all threads that have the same element and record rank.
        if (values[i] == values[tid]) {
            alive = false;
            ranks[tid] = count - 1;
        }
    }

    return count;
}


__device__ void sortColors(const float * values, int * ranks)
{
    const int tid = threadIdx.x;

    int rank = 0;

    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }
    
    ranks[tid] = rank;

    // Resolve elements with the same index.
    #pragma unroll
    for (int i = 0; i < 15; i++)
    {
        if ((tid > i) & (ranks[tid] == ranks[i])) ++ranks[tid];
    }
}

__device__ void sortColors(const float * values, int * ranks, int count)
{
    const int tid = threadIdx.x;

    int rank = 0;

    #pragma unroll
    for (int i = 0; i < count; i++)
    {
        rank += (values[i] < values[tid]);
    }
    
    ranks[tid] = rank;

    // Resolve elements with the same index.
    #pragma unroll
    for (int i = 0; i < count-1; i++)
    {
        if ((tid > i) & (ranks[tid] == ranks[i])) ++ranks[tid];
    }
}



////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////

__device__ void loadColorBlockTex(uint firstBlock, uint blockWidth, float3 colors[16], float3 sums[16], int xrefs[16], int * sameColor)
{
    const int bid = blockIdx.x;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    if (idx < 16)
    {
        float x = 4 * ((firstBlock + bid) % blockWidth) + idx % 4; // @@ Avoid mod and div by using 2D grid?
        float y = 4 * ((firstBlock + bid) / blockWidth) + idx / 4;

        // Read color and copy to shared mem.
        float4 c = tex2D(tex, x, y);

        colors[idx].x = c.z;
        colors[idx].y = c.y;
        colors[idx].z = c.x;

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float3 axis = bestFitLine(colors, sums[0], kColorMetric);

        *sameColor = (axis == make_float3(0, 0, 0));

        dps[idx] = dot(colors[idx], axis);

        sortColors(dps, xrefs);

        float3 tmp = colors[idx];
        colors[xrefs[idx]] = tmp;
    }
}

/*
__device__ void loadColorBlockTex(uint firstBlock, uint w, float3 colors[16], float3 sums[16], float weights[16], int xrefs[16], int * sameColor)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;

	__shared__ float dps[16];

	if (idx < 16)
	{
		float x = 4 * ((firstBlock + bid) % w) + idx % 4; // @@ Avoid mod and div by using 2D grid?
		float y = 4 * ((firstBlock + bid) / w) + idx / 4;

		// Read color and copy to shared mem.
		float4 c = tex2D(tex, x, y);

		colors[idx].x = c.z;
		colors[idx].y = c.y;
		colors[idx].z = c.x;
		weights[idx] = 1;

		int count = packColors(colors, weights);
		if (idx < count)
		{
			// Sort colors along the best fit line.
			colorSums(colors, sums);
			float3 axis = bestFitLine(colors, sums[0], kColorMetric);
			
			*sameColor = (axis == make_float3(0, 0, 0));
			
			dps[idx] = dot(colors[idx], axis);
			
			sortColors(dps, xrefs);
			
			float3 tmp = colors[idx];
			colors[xrefs[idx]] = tmp;
		}
	}
}
*/

__device__ void loadColorBlockTex(uint firstBlock, uint width, float3 colors[16], float3 sums[16], float weights[16], int xrefs[16], int * sameColor)
{
    const int bid = blockIdx.x;
    const int idx = threadIdx.x;

    __shared__ float3 rawColors[16];
    __shared__ float dps[16];

    if (idx < 16)
    {
        float x = 4 * ((firstBlock + bid) % width) + idx % 4; // @@ Avoid mod and div by using 2D grid?
        float y = 4 * ((firstBlock + bid) / width) + idx / 4;

        // Read color and copy to shared mem.
        float4 c = tex2D(tex, x, y);

        rawColors[idx].x = c.z;
        rawColors[idx].y = c.y;
        rawColors[idx].z = c.x;
        weights[idx] = c.w;

        colors[idx] = rawColors[idx] * weights[idx];

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float3 axis = bestFitLine(colors, sums[0], kColorMetric);

        *sameColor = (axis == make_float3(0, 0, 0));

        // Single color compressor needs unweighted colors.
        if (*sameColor) colors[idx] = rawColors[idx];

        dps[idx] = dot(colors[idx], axis);

        sortColors(dps, xrefs);

        float3 tmp = colors[idx];
        float w = weights[idx];
        colors[xrefs[idx]] = tmp;
        weights[xrefs[idx]] = w;
    }
}

__device__ void loadColorBlock(const uint * image, float2 colors[16], float2 sums[16], int xrefs[16], int * sameColor)
{
    const int bid = blockIdx.x;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    if (idx < 16)
    {
        // Read color and copy to shared mem.
        uint c = image[(bid) * 16 + idx];

        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float2 axis = bestFitLine(colors, sums[0]);

        *sameColor = (axis == make_float2(0, 0));

        dps[idx] = dot(colors[idx], axis);

        sortColors(dps, xrefs);

        float2 tmp = colors[idx];
        colors[xrefs[idx]] = tmp;
    }
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
__device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetricSqr);
}

__constant__ const float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ const float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const uint prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const uint prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

__device__ float evalPermutation4(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    uint akku = 0;

    // Compute alpha & beta for this permutation.
    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float(akku & 0xff);
    float3 betax_sum = 9.0f * color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    //float3 e = blockError4(colors, permutation, *start, *end);

    return (1.0f / 9.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation3(const float3 * colors, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    uint akku = 0;

    // Compute alpha & beta for this permutation.
    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float(akku & 0xff);
    float3 betax_sum = 4.0f * color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    //float3 e = blockError3(colors, permutation, a, b);

    return (1.0f / 4.0f) * dot(e, kColorMetricSqr);
}

__device__ float evalPermutation4(const float3 * colors, const float * weights, float3 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha * weights[i];
        beta2_sum += beta * beta * weights[i];
        alphabeta_sum += alpha * beta * weights[i];
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetricSqr);
}

/*
__device__ float evalPermutation3(const float3 * colors, const float * weights, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha * weights[i];
        beta2_sum += beta * beta * weights[i];
        alphabeta_sum += alpha * beta * weights[i];
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand565(a, start);
    b = roundAndExpand565(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetricSqr);
}
*/

__device__ float evalPermutation4(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float2 alphax_sum = make_float2(0.0f, 0.0f);
    uint akku = 0;

    // Compute alpha & beta for this permutation.
    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float(akku & 0xff);
    float2 betax_sum = 9.0f * color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6 color and expand...
    a = roundAndExpand56(a, start);
    b = roundAndExpand56(b, end);

    // compute the error
    float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (1.0f / 9.0f) * (e.x + e.y);
}

__device__ float evalPermutation3(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float2 alphax_sum = make_float2(0.0f, 0.0f);
    uint akku = 0;

    // Compute alpha & beta for this permutation.
    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float(akku & 0xff);
    float2 betax_sum = 4.0f * color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6 color and expand...
    a = roundAndExpand56(a, start);
    b = roundAndExpand56(b, end);

    // compute the error
    float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (1.0f / 4.0f) * (e.x + e.y);
}

__device__ float evalPermutationCTX(const float2 * colors, float2 color_sum, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float2 alphax_sum = make_float2(0.0f, 0.0f);
    uint akku = 0;

    // Compute alpha & beta for this permutation.
    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float(akku & 0xff);
    float2 betax_sum = 9.0f * color_sum - alphax_sum;

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float2 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float2 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 8-8 color and expand...
    a = roundAndExpand88(a, start);
    b = roundAndExpand88(b, end);

    // compute the error
    float2 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (1.0f / 9.0f) * (e.x + e.y);
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate all permutations
////////////////////////////////////////////////////////////////////////////////
__device__ void evalAllPermutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    __shared__ uint s_permutations[160];

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;

        ushort start, end;
        uint permutation = permutations[pidx];
        if (pidx < 160) s_permutations[pidx] = permutation;

        float error = evalPermutation4(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;	// Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 160) break;

        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;

            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}

/*
__device__ void evalAllPermutations(const float3 * colors, const float * weights, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
	const int idx = threadIdx.x;
	
	float bestError = FLT_MAX;
	
	__shared__ uint s_permutations[160];
	
	for(int i = 0; i < 16; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 992) break;
		
		ushort start, end;
		uint permutation = permutations[pidx];
		if (pidx < 160) s_permutations[pidx] = permutation;

		float error = evalPermutation4(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		int pidx = idx + NUM_THREADS * i;
		if (pidx >= 160) break;
		
		ushort start, end;
		uint permutation = s_permutations[pidx];
		float error = evalPermutation3(colors, weights, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}

	errors[idx] = bestError;
}
*/

__device__ void evalAllPermutations(const float2 * colors, float2 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    __shared__ uint s_permutations[160];

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;

        ushort start, end;
        uint permutation = permutations[pidx];
        if (pidx < 160) s_permutations[pidx] = permutation;

        float error = evalPermutation4(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;	// Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 160) break;

        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;

            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}

__device__ void evalLevel4Permutations(const float3 * colors, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;

        ushort start, end;
        uint permutation = permutations[pidx];

        float error = evalPermutation4(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;	// Flip indices.
    }

    errors[idx] = bestError;
}

__device__ void evalLevel4Permutations(const float3 * colors, const float * weights, float3 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;

        ushort start, end;
        uint permutation = permutations[pidx];

        float error = evalPermutation4(colors, weights, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;	// Flip indices.
    }

    errors[idx] = bestError;
}

__device__ void evalAllPermutationsCTX(const float2 * colors, float2 colorSum, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 704) break;

        ushort start, end;
        uint permutation = permutations[pidx];

        float error = evalPermutationCTX(colors, colorSum, permutation, &start, &end);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;	// Flip indices.
    }

    errors[idx] = bestError;
}


////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float * errors)
{
    const int idx = threadIdx.x;

    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

    for(int d = NUM_THREADS/2; d > 32; d >>= 1)
    {
        __syncthreads();

        if (idx < d)
        {
            float err0 = errors[idx];
            float err1 = errors[idx + d];

            if (err1 < err0) {
                errors[idx] = err1;
                indices[idx] = indices[idx + d];
            }
        }
    }

    __syncthreads();

    // unroll last 6 iterations
    if (idx < 32)
    {
        if (errors[idx + 32] < errors[idx]) {
            errors[idx] = errors[idx + 32];
            indices[idx] = indices[idx + 32];
        }
        if (errors[idx + 16] < errors[idx]) {
            errors[idx] = errors[idx + 16];
            indices[idx] = indices[idx + 16];
        }
        if (errors[idx + 8] < errors[idx]) {
            errors[idx] = errors[idx + 8];
            indices[idx] = indices[idx + 8];
        }
        if (errors[idx + 4] < errors[idx]) {
            errors[idx] = errors[idx + 4];
            indices[idx] = indices[idx + 4];
        }
        if (errors[idx + 2] < errors[idx]) {
            errors[idx] = errors[idx + 2];
            indices[idx] = indices[idx + 2];
        }
        if (errors[idx + 1] < errors[idx]) {
            errors[idx] = errors[idx + 1];
            indices[idx] = indices[idx + 1];
        }
    }

    __syncthreads();

    return indices[0];
}


////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
    const int bid = blockIdx.x;

    if (start == end)
    {
        permutation = 0;
    }

    // Reorder permutation.
    uint indices = 0;
    for(int i = 0; i < 16; i++)
    {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }

    // Write endpoints.
    result[bid].x = (end << 16) | start;

    // Write palette indices.
    result[bid].y = indices;
}

__device__ void saveBlockDXT1_Parallel(uint endpoints, float3 colors[16], int xrefs[16], uint * result)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid < 16)
    {
        int3 color = float3ToInt3(colors[xrefs[tid]]);

        ushort endpoint0 = endpoints & 0xFFFF;
        ushort endpoint1 = endpoints >> 16;

        int3 palette[4];
        palette[0] = color16ToInt3(endpoint0);
        palette[1] = color16ToInt3(endpoint1);

        int d0 = colorDistance(palette[0], color);
        int d1 = colorDistance(palette[1], color);

        uint index;
        if (endpoint0 > endpoint1) 
        {
            palette[2].x = (2 * palette[0].x + palette[1].x) / 3;
            palette[2].y = (2 * palette[0].y + palette[1].y) / 3;
            palette[2].z = (2 * palette[0].z + palette[1].z) / 3;

            palette[3].x = (2 * palette[1].x + palette[0].x) / 3;
            palette[3].y = (2 * palette[1].y + palette[0].y) / 3;
            palette[3].z = (2 * palette[1].z + palette[0].z) / 3;
            
            int d2 = colorDistance(palette[2], color);
            int d3 = colorDistance(palette[3], color);

            // Compute the index that best fit color.
            uint b0 = d0 > d3;
            uint b1 = d1 > d2;
            uint b2 = d0 > d2;
            uint b3 = d1 > d3;
            uint b4 = d2 > d3;

            uint x0 = b1 & b2;
            uint x1 = b0 & b3;
            uint x2 = b0 & b4;

            index = (x2 | ((x0 | x1) << 1));
        }
        else {
            palette[2].x = (palette[0].x + palette[1].x) / 2;
            palette[2].y = (palette[0].y + palette[1].y) / 2;
            palette[2].z = (palette[0].z + palette[1].z) / 2;

            int d2 = colorDistance(palette[2], color);

            index = 0;
            if (d1 < d0 && d1 < d2) index = 1;
            else if (d2 < d0) index = 2;
        }

        __shared__ uint indices[16];

        indices[tid] = index << (2 * tid);
        if (tid < 8) indices[tid] |= indices[tid+8];
        if (tid < 4) indices[tid] |= indices[tid+4];
        if (tid < 2) indices[tid] |= indices[tid+2];
        if (tid < 1) indices[tid] |= indices[tid+1];

        if (tid < 2) {
            result[2 * bid + tid] = tid == 0 ? endpoints : indices[0];
        }
    }
}

__device__ void saveBlockDXT1_Parallel(uint endpoints, uint permutation, int xrefs[16], uint * result)
{
    const int tid = threadIdx.x;    
    const int bid = blockIdx.x;

    if (tid < 16)
    {
        // Reorder permutation.
        uint index = ((permutation >> (2 * xrefs[tid])) & 3) << (2 * tid);
        __shared__ uint indices[16];

        indices[tid] = index;
        if (tid < 8) indices[tid] |= indices[tid+8];
        if (tid < 4) indices[tid] |= indices[tid+4];
        if (tid < 2) indices[tid] |= indices[tid+2];
        if (tid < 1) indices[tid] |= indices[tid+1];
    	
        if (tid < 2) {
            result[2 * bid + tid] = tid == 0 ? endpoints : indices[0];
        }
    }
}


__device__ void saveBlockCTX1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result)
{
    saveBlockDXT1(start, end, permutation, xrefs, result);
}

__device__ void saveSingleColorBlockDXT1(float3 color, uint2 * result)
{
    const int bid = blockIdx.x;

    int r = color.x * 255;
    int g = color.y * 255;
    int b = color.z * 255;

    ushort color0 = (OMatch5[r][0] << 11) | (OMatch6[g][0] << 5) | OMatch5[b][0];
    ushort color1 = (OMatch5[r][1] << 11) | (OMatch6[g][1] << 5) | OMatch5[b][1];

    if (color0 < color1)
    {
        result[bid].x = (color0 << 16) | color1;
        result[bid].y = 0xffffffff;
    }
    else
    {
        result[bid].x = (color1 << 16) | color0;
        result[bid].y = 0xaaaaaaaa;
    }
}

__device__ void saveSingleColorBlockDXT1(float2 color, uint2 * result)
{
    const int bid = blockIdx.x;

    int r = color.x * 255;
    int g = color.y * 255;

    ushort color0 = (OMatch5[r][0] << 11) | (OMatch6[g][0] << 5);
    ushort color1 = (OMatch5[r][1] << 11) | (OMatch6[g][1] << 5);

    if (color0 < color1)
    {
        result[bid].x = (color0 << 16) | color1;
        result[bid].y = 0xffffffff;
    }
    else
    {
        result[bid].x = (color1 << 16) | color0;
        result[bid].y = 0xaaaaaaaa;
    }
}

__device__ void saveSingleColorBlockCTX1(float2 color, uint2 * result)
{
    const int bid = blockIdx.x;

    int r = color.x * 255;
    int g = color.y * 255;

    ushort color0 = (r << 8) | (g);

    result[bid].x = (color0 << 16) | color0;
    result[bid].y = 0x00000000;
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////

__global__ void compressDXT1(uint firstBlock, uint blockWidth, const uint * permutations, uint2 * result)
{
    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];
    __shared__ int sameColor;

    loadColorBlockTex(firstBlock, blockWidth, colors, sums, xrefs, &sameColor);

    __syncthreads();

    if (sameColor)
    {
        if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
        return;
    }

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];
    evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
    
    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);

    __shared__ uint s_bestEndPoints;
    //__shared__ uint s_bestPermutation;

    // Only write the result of the winner thread.
    if (threadIdx.x == minIdx)
    {
        s_bestEndPoints = (bestEnd << 16) | bestStart;
        //s_bestPermutation = (bestStart != bestEnd) ? bestPermutation : 0;
    }

    __syncthreads();

    saveBlockDXT1_Parallel(s_bestEndPoints, colors, xrefs, (uint *)result);
    //saveBlockDXT1_Parallel(s_bestEndPoints, s_bestPermutation, xrefs, (uint *)result);
}


__global__ void compressLevel4DXT1(uint firstBlock, uint blockWidth, const uint * permutations, uint2 * result)
{
    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];
    __shared__ int sameColor;

    loadColorBlockTex(firstBlock, blockWidth, colors, sums, xrefs, &sameColor);

    __syncthreads();

    if (sameColor)
    {
        if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
        return;
    }

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalLevel4Permutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);

    // Only write the result of the winner thread.
    if (threadIdx.x == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
    }
}

__global__ void compressWeightedDXT1(uint firstBlock, uint blockWidth, const uint * permutations, uint2 * result)
{
    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ float weights[16];
    __shared__ int xrefs[16];
    __shared__ int sameColor;

    loadColorBlockTex(firstBlock, blockWidth, colors, sums, weights, xrefs, &sameColor);

    __syncthreads();

    if (sameColor)
    {
        if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
        return;
    }

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);

    // Use a parallel reduction to find minimum error.
    int minIdx = findMinError(errors);

    // Only write the result of the winner thread.
    if (threadIdx.x == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
    }
}


__global__ void compressNormalDXT1(const uint * permutations, const uint * image, uint2 * result)
{
    __shared__ float2 colors[16];
    __shared__ float2 sums[16];
    __shared__ int xrefs[16];
    __shared__ int sameColor;

    loadColorBlock(image, colors, sums, xrefs, &sameColor);

    __syncthreads();

    if (sameColor)
    {
        if (threadIdx.x == 0) saveSingleColorBlockDXT1(colors[0], result);
        return;
    }

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);

    // Only write the result of the winner thread.
    if (threadIdx.x == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result);
    }
}

__global__ void compressCTX1(const uint * permutations, const uint * image, uint2 * result)
{
    __shared__ float2 colors[16];
    __shared__ float2 sums[16];
    __shared__ int xrefs[16];
    __shared__ int sameColor;

    loadColorBlock(image, colors, sums, xrefs, &sameColor);

    __syncthreads();

    if (sameColor)
    {
        if (threadIdx.x == 0) saveSingleColorBlockCTX1(colors[0], result);
        return;
    }

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutationsCTX(colors, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);

    // Only write the result of the winner thread.
    if (threadIdx.x == minIdx)
    {
        saveBlockCTX1(bestStart, bestEnd, bestPermutation, xrefs, result);
    }
}


/*
__device__ float computeError(const float weights[16], uchar a0, uchar a1)
{
	float palette[6];
	palette[0] = (6.0f/7.0f * a0 + 1.0f/7.0f * a1);
	palette[1] = (5.0f/7.0f * a0 + 2.0f/7.0f * a1);
	palette[2] = (4.0f/7.0f * a0 + 3.0f/7.0f * a1);
	palette[3] = (3.0f/7.0f * a0 + 4.0f/7.0f * a1);
	palette[4] = (2.0f/7.0f * a0 + 5.0f/7.0f * a1);
	palette[5] = (1.0f/7.0f * a0 + 6.0f/7.0f * a1);

	float total = 0.0f;

	for (uint i = 0; i < 16; i++)
	{
		float alpha = weights[i];

		float error = a0 - alpha;
		error = min(error, palette[0] - alpha);
		error = min(error, palette[1] - alpha);
		error = min(error, palette[2] - alpha);
		error = min(error, palette[3] - alpha);
		error = min(error, palette[4] - alpha);
		error = min(error, palette[5] - alpha);
		error = min(error, a1 - alpha);
		
		total += error;
	}
	
	return total;
}

inline __device__ uchar roundAndExpand(float a)
{
	return rintf(__saturatef(a) * 255.0f);
}
*/
/*
__device__ void optimizeAlpha8(const float alphas[16], uchar & a0, uchar & a1)
{
	float alpha2_sum = 0;
	float beta2_sum = 0;
	float alphabeta_sum = 0;
	float alphax_sum = 0;
	float betax_sum = 0;

	for (int i = 0; i < 16; i++)
	{
		uint idx = index[i];
		float alpha;
		if (idx < 2) alpha = 1.0f - idx;
		else alpha = (8.0f - idx) / 7.0f;
		
		float beta = 1 - alpha;

		alpha2_sum += alpha * alpha;
		beta2_sum += beta * beta;
		alphabeta_sum += alpha * beta;
		alphax_sum += alpha * alphas[i];
		betax_sum += beta * alphas[i];
	}

	const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

	float a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
	float b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

	a0 = roundAndExpand8(a);
	a1 = roundAndExpand8(b);
}
*/
/*
__device__ void compressAlpha(const float alphas[16], uint4 * result)
{
	const int tid = threadIdx.x;
	
	// Compress alpha block!
	// Brute force approach:
	// Try all color pairs: 256*256/2 = 32768, 32768/64 = 512 iterations?

	// Determine min & max alphas

	float A0, A1;

	if (tid < 16)
	{
		__shared__ uint s_alphas[16];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = min(s_alphas[tid], s_alphas[tid^1]);
		A0 = s_alphas[tid];
		
		s_alphas[tid] = alphas[tid];
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^8]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^4]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^2]);
		s_alphas[tid] = max(s_alphas[tid], s_alphas[tid^1]);
		A1 = s_alphas[tid];
	}

	__syncthreads();

	int minIdx = 0;

	if (A1 - A0 > 8)
	{
		float bestError = FLT_MAX;

		// 64 threads -> 8x8
		// divide [A1-A0] in partitions.
		// test endpoints 
		
		for (int i = 0; i < 128; i++)
		{
			uint idx = (i * NUM_THREADS + tid) * 4;
			uchar a0 = idx & 255;
			uchar a1 = idx >> 8;
			
			float error = computeError(alphas, a0, a1);
			
			if (error < bestError)
			{
				bestError = error;
				A0 = a0;
				A1 = a1;
			}
		}
		
		__shared__ float errors[NUM_THREADS];
		errors[tid] = bestError;
		
		// Minimize error.
		minIdx = findMinError(errors);

	}

	if (minIdx == tid)
	{
		// @@ Compute indices.
	
		// @@ Write alpha block.
	}
}

__global__ void compressDXT5(const uint * permutations, const uint * image, uint4 * result)
{
	__shared__ float3 colors[16];
	__shared__ float3 sums[16];
	__shared__ float weights[16];
	__shared__ int xrefs[16];
	
	loadColorBlock(image, colors, sums, weights, xrefs);
	
	__syncthreads();

	compressAlpha(weights, result);	

	ushort bestStart, bestEnd;
	uint bestPermutation;

	__shared__ float errors[NUM_THREADS];
	
	evalLevel4Permutations(colors, weights, sums[0], permutations, bestStart, bestEnd, bestPermutation, errors);
	
	// Use a parallel reduction to find minimum error.
	int minIdx = findMinError(errors);
	
	// Only write the result of the winner thread.
	if (threadIdx.x == minIdx)
	{
		saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, (uint2 *)result);
	}
}
*/

/*__device__ void evaluatePalette(uint alpha0, uint alpha1, uint alphas[8])
{
	alpha[0] = alpha0;
	alpha[1] = alpha1;
	alpha[2] = (6 * alpha[0] + 1 * alpha[1]) / 7;	// bit code 010
	alpha[3] = (5 * alpha[0] + 2 * alpha[1]) / 7;	// bit code 011
	alpha[4] = (4 * alpha[0] + 3 * alpha[1]) / 7;	// bit code 100
	alpha[5] = (3 * alpha[0] + 4 * alpha[1]) / 7;	// bit code 101
	alpha[6] = (2 * alpha[0] + 5 * alpha[1]) / 7;	// bit code 110
	alpha[7] = (1 * alpha[0] + 6 * alpha[1]) / 7;	// bit code 111
}

__device__ uint computeAlphaError(const uint block[16], uint alpha0, uint alpha1, int bestError = INT_MAX)
{
	uint8 alphas[8];
	evaluatePalette(alpha0, alpha1, alphas);

	int totalError = 0;

	for (uint i = 0; i < 16; i++)
	{
		uint8 alpha = block[i];

		// @@ It should be possible to do this much faster.

		int minDist = INT_MAX;
		for (uint p = 0; p < 8; p++)
		{
			int dist = alphaDistance(alpha, alphas[p]);
			minDist = min(dist, minDist);
		}



		totalError += minDist;

		if (totalError > bestError)
		{
			// early out
			return totalError;
		}
	}

	return totalError;
}


void compressDXT5A(uint alpha[16])
{
	// Get min/max alpha.
	for (uint i = 0; i < 16; i++)
	{
		mina = min(mina, alpha[i]);
		maxa = max(maxa, alpha[i]);
	}

	dxtBlock->alpha0 = maxa;
	dxtBlock->alpha1 = mina;

	if (maxa - mina > 8)
	{
		int besterror = computeAlphaError(rgba, dxtBlock);
		int besta0 = maxa;
		int besta1 = mina;

		// Expand search space a bit.
		const int alphaExpand = 8;
		mina = (mina <= alphaExpand) ? 0 : mina - alphaExpand;
		maxa = (maxa <= 255-alphaExpand) ? 255 : maxa + alphaExpand;

		for (int a0 = mina+9; a0 < maxa; a0++)
		{
			for (int a1 = mina; a1 < a0-8; a1++)
			{
				nvDebugCheck(a0 - a1 > 8);

				dxtBlock->alpha0 = a0;
				dxtBlock->alpha1 = a1;
				int error = computeAlphaError(rgba, dxtBlock, besterror);

				if (error < besterror)
				{
					besterror = error;
					besta0 = a0;
					besta1 = a1;
				}
			}
		}

		dxtBlock->alpha0 = besta0;
		dxtBlock->alpha1 = besta1;
	}
}

__global__ void compressDXT5n(uint blockNum, uint2 * d_result)
{
	uint idx = blockIdx.x * 128 + threadIdx.x;

	if (idx >= blockNum)
	{
		return;
	}

	// @@ Ideally we would load the data to shared mem to achieve coalesced global mem access.
	// @@ Blocks would require too much shared memory (8k) and limit occupancy.

	// @@ Ideally we should use SIMD processing, multiple threads (4-8) processing the same block.
	// That simplifies coalescing, and reduces divergence.

	// @@ Experiment with texture. That's probably the most simple approach.

	uint x[16];
	uint y[16];


}
*/


////////////////////////////////////////////////////////////////////////////////
// Setup kernel
////////////////////////////////////////////////////////////////////////////////

extern "C" void setupOMatchTables(const void * OMatch5Src, size_t OMatch5Size, const void * OMatch6Src, size_t OMatch6Size)
{
    // Init single color lookup contant tables.
    hipMemcpyToSymbol(HIP_SYMBOL(OMatch5), OMatch5Src, OMatch5Size, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(OMatch6), OMatch6Src, OMatch6Size, 0, hipMemcpyHostToDevice);
}

extern "C" void setupCompressKernel(const float weights[3])
{
    // Set constants.
    hipMemcpyToSymbol(HIP_SYMBOL(kColorMetric), weights, sizeof(float) * 3, 0);

    float weightsSqr[3];
    weightsSqr[0] = weights[0] * weights[0];
    weightsSqr[1] = weights[1] * weights[1];
    weightsSqr[2] = weights[2] * weights[2];

    hipMemcpyToSymbol(HIP_SYMBOL(kColorMetricSqr), weightsSqr, sizeof(float) * 3, 0);
}

extern "C" void bindTextureToArray(hipArray * d_data)
{
    // Setup texture
    tex.normalized = false;
    tex.filterMode = hipFilterModePoint;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;

    hipBindTextureToArray(tex, d_data);
}



////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////

// DXT1 compressors:
extern "C" void compressKernelDXT1(uint firstBlock, uint blockNum, uint blockWidth, uint * d_result, uint * d_bitmaps)
{
    compressDXT1<<<blockNum, NUM_THREADS>>>(firstBlock, blockWidth, d_bitmaps, (uint2 *)d_result);
}

extern "C" void compressKernelDXT1_Level4(uint firstBlock, uint blockNum, uint blockWidth, uint * d_result, uint * d_bitmaps)
{
    compressLevel4DXT1<<<blockNum, NUM_THREADS>>>(firstBlock, blockWidth, d_bitmaps, (uint2 *)d_result);
}

extern "C" void compressWeightedKernelDXT1(uint firstBlock, uint blockNum, uint blockWidth, uint * d_result, uint * d_bitmaps)
{
    compressWeightedDXT1<<<blockNum, NUM_THREADS>>>(firstBlock, blockWidth, d_bitmaps, (uint2 *)d_result);
}

// @@ DXT1a compressors.


// @@ DXT3 compressors:
extern "C" void compressKernelDXT3(uint firstBlock, uint blockNum, uint blockWidth, uint * d_result, uint * d_bitmaps)
{
    //compressDXT3<<<blockNum, NUM_THREADS>>>(firstBlock, blockWidth, d_bitmaps, (uint2 *)d_result);
}

extern "C" void compressWeightedKernelDXT3(uint firstBlock, uint blockNum, uint blockWidth, uint * d_result, uint * d_bitmaps)
{
    //compressWeightedDXT3<<<blockNum, NUM_THREADS>>>(firstBlock, blockWidth, d_bitmaps, (uint2 *)d_result);
}


// @@ DXT5 compressors.
extern "C" void compressKernelDXT5(uint firstBlock, uint blockNum, uint w, uint * d_result, uint * d_bitmaps)
{
    //compressDXT5<<<blockNum, NUM_THREADS>>>(firstBlock, w, d_bitmaps, (uint2 *)d_result);
}

extern "C" void compressWeightedKernelDXT5(uint firstBlock, uint blockNum, uint w, uint * d_result, uint * d_bitmaps)
{
    //compressWeightedDXT5<<<blockNum, NUM_THREADS>>>(firstBlock, w, d_bitmaps, (uint2 *)d_result);
}





/*
extern "C" void compressNormalKernelDXT1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
    compressNormalDXT1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}

extern "C" void compressKernelCTX1(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps)
{
    compressCTX1<<<blockNum, NUM_THREADS>>>(d_bitmaps, d_data, (uint2 *)d_result);
}
*/
/*
extern "C" void compressKernelDXT5n(uint blockNum, hipArray * d_data, uint * d_result)
{
//    compressDXT5n<<<blockNum/128, 128>>>(blockNum, (uint2 *)d_result);
}
*/
